#include "hip/hip_runtime.h"
#include "tracer.h"
#include "ray.h"
#include "global.h"
#include "vector.h"
#include "consts.h"

#include "gpu/ray_gpu.cu"
#include "gpu/vector_gpu.cu" 
    
#include <hip/hip_runtime.h>  
#include <vector>
#include <assert.h> 
using namespace std; 
 
#include "gpu/ray_gpu.cu"
#include "gpu_util.h"
#include "gpu/geometry_gpu.cu"

BBox Tracer::_bbox;

////	Color Map Params

__device__
float gpuKnotValues[5] = {0};
__device__
int gpuKnotColors[5] = {0};

////
__device__ float tex_fStart = 0;
__device__ float tex_fEnd = 0;
__device__ int tex_Width = 0;
__device__ int tex_Height = 0;
__device__ float *tex_data = NULL;
__device__ int nDevPlaneSampleCount;

__device__ PrimGpuObj *pCap0Device_dev = NULL;
__device__ unsigned nCap0TriCount_dev = 0;

__device__ PrimGpuObj *pCap1Device_dev = NULL;
__device__ unsigned nCap1TriCount_dev = 0;

__device__ PrimGpuObj *pSliceDevice_dev = NULL;
__device__ unsigned nSliceTriCount_dev = 0;

__device__ PrimGpuObj *pPlaneDevice_dev = NULL;
__device__ unsigned nPlaneTriCount_dev = 0;

__global__ 
void _setTexInfo(float fStart, float fEnd, int texWidth, int texHeight, float *deviceTex, int nPlaneSampleCount, 
				 PrimGpuObj *pCap0, unsigned nCap0TriCount, PrimGpuObj *pCap1, unsigned nCap1TriCount,
				 PrimGpuObj *pSlice, unsigned nSliceTriCount,
				 float knotValue0, float knotValue1, float knotValue2, float knotValue3, float knotValue4, 
				 int knotColor0, int knotColor1, int knotColor2, int knotColor3, int knotColor4,
				 PrimGpuObj *pPlane, unsigned nPlaneTriCount)
{
	tex_fStart = fStart;
	tex_fEnd = fEnd;

	tex_Width = texWidth;
	tex_Height = texHeight;

	tex_data = deviceTex;

	nDevPlaneSampleCount = nPlaneSampleCount;

	pCap0Device_dev = pCap0;
	nCap0TriCount_dev = nCap0TriCount;

	pCap1Device_dev = pCap1;
	nCap1TriCount_dev = nCap1TriCount;

	pSliceDevice_dev = pSlice;
	nSliceTriCount_dev = nSliceTriCount;

	pPlaneDevice_dev = pPlane;
	nPlaneTriCount_dev = nPlaneTriCount;
	
	//
	gpuKnotValues[0] = knotValue0;
	gpuKnotValues[1] = knotValue1;
	gpuKnotValues[2] = knotValue2;
	gpuKnotValues[3] = knotValue3;
	gpuKnotValues[4] = knotValue4;

	gpuKnotColors[0] = knotColor0;
	gpuKnotColors[1] = knotColor1;
	gpuKnotColors[2] = knotColor2;
	gpuKnotColors[3] = knotColor3;
	gpuKnotColors[4] = knotColor4;
}

////
__device__
void clampColor_gpu(float *pColor)
{
	pColor[0] = pColor[0] > 1.f ? 1.f : (pColor[0] < 0.f ? 0.f : pColor[0]);
	pColor[1] = pColor[1] > 1.f ? 1.f : (pColor[1] < 0.f ? 0.f : pColor[1]);
	pColor[2] = pColor[2] > 1.f ? 1.f : (pColor[2] < 0.f ? 0.f : pColor[2]);
}

__device__
unsigned getThreadInx()
{
	return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__
void genViewRay(Ray_gpu &ray, unsigned row, unsigned col, unsigned nWidth, unsigned nHeight,
						float fViewPlaneRatio,
						float eye_x, float eye_y, float eye_z, 
						float viewPlaneCtr_x, float viewPlaneCtr_y, float viewPlaneCtr_z, 
						float _rightVec_x, float _rightVec_y, float _rightVec_z, 
						float _upVec_x, float _upVec_y, float _upVec_z, float rdm[2], float fSamplingDeltaFactor)
{

	//	to fine the current primary ray starting point
	//
	vect3d_gpu nCurrCtr(viewPlaneCtr_x, viewPlaneCtr_y, viewPlaneCtr_z);
	
	//	right vec first
	vect3d_gpu rightVec(_rightVec_x, _rightVec_y, _rightVec_z);
	vecScale_gpu(rightVec, (col - nWidth/2.f) * fViewPlaneRatio / vecLen_gpu(&rightVec), rightVec);

	//	up vec second
	vect3d_gpu upVec(_upVec_x, _upVec_y, _upVec_z);
	vecScale_gpu(upVec, (row - nHeight/2.f) * fViewPlaneRatio/ vecLen_gpu(&upVec), upVec);
	
	point2point_gpu(nCurrCtr, rightVec, nCurrCtr);
	point2point_gpu(nCurrCtr, upVec, nCurrCtr);

	//	Randomize
	vect3d_gpu vDeltaXVec;
	vecScale_gpu(rightVec, fViewPlaneRatio * fSamplingDeltaFactor * rdm[0], vDeltaXVec);

	vect3d_gpu vDeltaYVec;
	vecScale_gpu(upVec, fViewPlaneRatio * fSamplingDeltaFactor * rdm[1], vDeltaYVec);

	point2point_gpu(nCurrCtr, vDeltaXVec, nCurrCtr);			
	point2point_gpu(nCurrCtr, vDeltaYVec, nCurrCtr);	
	
	//	
	vect3d_gpu eye(eye_x, eye_y, eye_z);
	vect3d_gpu viewDir;
	points2vec_gpu(eye, nCurrCtr, viewDir);
	normalize_gpu(viewDir);
	
	vecCopy_gpu(ray.start_point, nCurrCtr);
	vecCopy_gpu(ray.direction_vec, viewDir);
}

__device__
void getCellInx(PrimGpuObj *pObj, PrimGpuObj *gpuObjs, float *hitPoint, 
				unsigned max_x, unsigned max_y, unsigned max_z,
				int inx[3])
{
	inx[0] = hitPoint[0] + max_x * 1.f / 2.f;	inx[0] = inx[0] >= max_x ? (max_x - 1) : (inx[0] < 0.f ? 0.f : inx[0]);
	inx[1] = hitPoint[1] + max_y * 1.f / 2.f;	inx[1] = inx[1] >= max_y ? (max_y - 1) : (inx[1] < 0.f ? 0.f : inx[1]);
	inx[2] = hitPoint[2] + max_z * 1.f / 2.f;	inx[2] = inx[2] >= max_z ? (max_z - 1) : (inx[2] < 0.f ? 0.f : inx[2]);
}

__device__
float getElecCellValue(int x, int y, int z, float *elecData, int *idData, int mark[4], bool cutHalf = false)
{
	if( x < 0 || x >= VOL_X || 
		y < 0 || y >= VOL_Y ||
		z < 0 || z >= VOL_Z )	// Hard-code it for now
	{
		return 0;
	}
#ifndef DATA_2D
	if(cutHalf && y >= (VOL_Y/2))
#else
	if(cutHalf && z >= (VOL_Z/2))
#endif
	{
		return 0;
	}
	
	unsigned offset = x + y * VOL_X + z * VOL_X * VOL_Y;
	
	//	ID selection

	int currId = *(idData + offset);
	if( mark[currId - 1] == 0 )
	{
		return 0;
	}

	return *(elecData + offset);
}

__device__
float lerp(float v1, float ratio1, float v2)
{
#ifdef DATA_2D
	v1 = (v1 == INVALID_VOLVAL) ? 0 : v1;
	v2 = (v2 == INVALID_VOLVAL) ? 0 : v2;
#endif
	return v1 * (1.f - ratio1) + v2 * ratio1;
}

__device__
float getHermiteValue(float x, float fP0_val, float fP0_der, float fP1_val, float fP1_der)
{
	float x2 = x * x;
	float x3 = x * x * x;

	float p0 = 2 * x3 - 3 * x2 + 1;
	float p1 = -2 * x3 + 3 * x2;
	float p2 = x3 - 2 * x2 + x;
	float p3 = x3 - x2;

	return (fP0_val * p0 + fP1_val * p1 + fP0_der * p2 + fP1_der * p3);
}

/*
 *
 		0, "White"
		1, "Black"
		2, "Red"
		3, "Orange"
		4, "Yellow"
		5, "Green"
		6, "Cyan"
		7, "Blue"
		8, "Purple"
		9, "Gray"
 */
#define WHITE {1,1,1}
#define BLACK {0,0,0}
#define RED {1,0,0}
#define ORANGE {1,0.647,0}
#define YELLOW {1,1,0}
#define GREEN {0,1,0}
#define CYAN {0,1,1}
#define BLUE {0,0,1}
#define PURPLE {0.62745,0.12549,0.941176}
#define GRAY {0.8,0.8,0.8}

__device__
void custom_color_map(float value, float *pCurrPix, 
					  float knotValue[5], 
						int knotColor[5],
						 int mMode, float *deviceTexData, int texWidth, int texHeight, float fStart, float fEnd)
{
	if(mMode == 0)	//	Value-based
	{
		float _colors[10][3] = {
								WHITE,
								BLACK,
								RED,
								ORANGE,
								YELLOW,
								GREEN,
								CYAN,
								BLUE,
								PURPLE,
								GRAY
							  };
		//	Find starting inx & delta
		int inx0 = -1;
		float delta = 0;
		for(int i = 1; i < 5; i ++)
		{
			if( value < knotValue[i])
			{
				inx0 = i - 1;
				delta = (value - knotValue[i - 1]) / (knotValue[i] - knotValue[i - 1]);
				break;
			}
		}

		//	return color
		//
		if(inx0 >= 0 && inx0 <= 4)	// should fall into the range
		{
			*(pCurrPix + 0) = (1 - delta) * _colors[knotColor[inx0]][0] + delta * _colors[knotColor[inx0 + 1]][0];
			*(pCurrPix + 1) = (1 - delta) * _colors[knotColor[inx0]][1] + delta * _colors[knotColor[inx0 + 1]][1];
			*(pCurrPix + 2) = (1 - delta) * _colors[knotColor[inx0]][2] + delta * _colors[knotColor[inx0 + 1]][2];
		}
	}
	else	//	Picture based
	{
		//	Texture Image should be horizontal
		//
		if(value < fStart || value > fEnd)
		{
			*(pCurrPix + 0) = AMBI_X;
			*(pCurrPix + 1) = AMBI_Y;
			*(pCurrPix + 2) = AMBI_Z;
		}
		else
		{
			int offset = (value - fStart) / (fEnd - fStart) * texWidth;
			offset = offset > texWidth ? texWidth  : offset;

			//	Pick color from the mid
			float *pDataStart = deviceTexData + texHeight / 2 * 3 * texWidth;

			float *pColor = pDataStart + 3 * offset;
			*(pCurrPix + 0) = *(pColor + 0);
			*(pCurrPix + 1) = *(pColor + 1);
			*(pCurrPix + 2) = *(pColor + 2);
		}
	}
}

#ifdef DATA_2D

__device__
float getSampleValueByBilinear(float point[3], int maxx, int maxy, int maxz, float *elecData,
							   int *idData, int mark[4])
{
	int x_inx = fabs(point[0]);
	int y_inx = fabs(point[1]);
	int x_inx_p = x_inx < maxx ? (x_inx + 1): x_inx;
	int y_inx_p = y_inx < maxy ? (y_inx + 1): y_inx;

	float v0 = getElecCellValue(x_inx, y_inx, -1, elecData, idData, mark, false);
	float v1 = getElecCellValue(x_inx_p, y_inx, -1, elecData, idData, mark, false);;
	float v2 = getElecCellValue(x_inx, y_inx_p, -1, elecData, idData, mark, false);;
	float v3 = getElecCellValue(x_inx_p, y_inx_p, -1, elecData, idData, mark, false);;

	return lerp(lerp(v0, 0.5, v1), 0.5, lerp(v2, 0.5, v3));
}

#endif

///	Ray-marching
///
__device__ 
float getSampleValueByTrilinear(	Ray_gpu &ray, float point[3], int maxx, int maxy, int maxz, 
										float *elecData, int *idData, int mark[4], int *layer = NULL, bool bShowPlane = false )
{
	//	Distance from Ray to Cell Center
	int inx[3] = {	(int)(point[0] + (VOL_X/2)),
					(int)(point[1] + (VOL_Y/2)),
					(int)(point[2] + (VOL_Z/2)) };

#ifdef DATA_2D
	if( getElecCellValue(inx[0], inx[1], inx[2], elecData, idData, mark, bShowPlane) == INVALID_VOLVAL)
	{
		return INVALID_VOLVAL;
	}
#endif

	if( inx[0] >= 0 && inx[0] < VOL_X &&
		inx[1] >= 0 && inx[1] < VOL_Y &&
		inx[2] >= 0 && inx[2] < VOL_Z) 
	{

		vect3d_gpu ctr( inx[0] - (VOL_X/2) + 0.5, 
						inx[1] - (VOL_Y/2) + 0.5, 
						inx[2] - (VOL_Z/2) + 0.5 );

#if 1
		//	Some pPoint is out of the current cell (what the hell?!)
		//	so check this pPoint then
		//
		float delta[3] = {  (point[0] - ctr.data[0]), 
							(point[1] - ctr.data[1]),
							(point[2] - ctr.data[2]) };
		for(int i = 0; i < 3; i ++)
		{
			delta[i] = delta[i] > 1.f ? 1.f : delta[i];
			delta[i] = delta[i] <-1.f ?-1.f : delta[i];
		}
			
		//	Tri-linear
		int sign[3] = { delta[0] > 0 ? 1 : -1, 
						delta[1] > 0 ? 1 : -1, 
						delta[2] > 0 ? 1 : -1 };

		//	4 x
		float x_ny_nz = lerp(getElecCellValue(inx[0],           inx[1],           inx[2], elecData, idData, mark, bShowPlane) ,  delta[0] * sign[0], 
							 getElecCellValue(inx[0] + sign[0], inx[1],           inx[2], elecData, idData, mark, bShowPlane));
		float x_fy_nz = lerp(getElecCellValue(inx[0],           inx[1] + sign[1], inx[2], elecData, idData, mark, bShowPlane) ,  delta[0] * sign[0], 
							 getElecCellValue(inx[0] + sign[0], inx[1] + sign[1], inx[2], elecData, idData, mark, bShowPlane));

		float x_ny_fz = lerp(getElecCellValue(inx[0],           inx[1],           inx[2] + sign[2], elecData, idData, mark, bShowPlane) ,  delta[0] * sign[0], 
							 getElecCellValue(inx[0] + sign[0], inx[1],           inx[2] + sign[2], elecData, idData, mark, bShowPlane));
		float x_fy_fz = lerp(getElecCellValue(inx[0],           inx[1] + sign[1], inx[2] + sign[2], elecData, idData, mark, bShowPlane) ,  delta[0] * sign[0], 
							 getElecCellValue(inx[0] + sign[0], inx[1] + sign[1], inx[2] + sign[2], elecData, idData, mark, bShowPlane));

		float y_nz = lerp(x_ny_nz, delta[1] * sign[1], x_fy_nz);
		float y_fz = lerp(x_ny_fz, delta[1] * sign[1], x_fy_fz);

		if(layer)
		{
			unsigned offset = inx[0] + inx[1] * VOL_X + inx[2] * VOL_X * VOL_Y;
			*layer = *(idData + offset);
		}

		return lerp(y_nz, delta[2] * sign[2], y_fz);

#else
		if(layer)
		{
			unsigned offset = inx[0] + inx[1] * VOL_X + inx[2] * VOL_X * VOL_Y;
			*layer = *(idData + offset);
		}
		return getElecCellValue(inx[0],           inx[1],           inx[2], elecData, idData, mark, bShowPlane);
#endif
	}

	return 0;
}

__device__
float ray_marching(	Ray_gpu &ray,
					float *ret, int *nCount, float start_point[3], float end_point[3], int max[3], float *elecData, 
					int tf_mode, float fP0_val, float fP0_der, float fP1_val, float fP1_der, int bShowGeo, int &bInGeo,
					bool bClipPlaneEnabled, float planeCtr0, float planeCtr1, float planeCtr2, float planeNorm0, float planeNorm1, float planeNorm2,
					int *idData, int id0, int id1, int id2, int id3, int bShowSlice, bool bShowPlane = false)
{
	float fStep = 1;	// TODO: to be passed in

	//	Get total marching step len
	//
	vect3d_gpu vTotalVec;
	points2vec_gpu(start_point, end_point, vTotalVec);
	float fTotalLen = vecLen_gpu(&vTotalVec);

	//	Get Stepping Vector
	//
	vect3d_gpu point(start_point[0], start_point[1], start_point[2]);
	normalize_gpu(ray.direction_vec);
	vect3d_gpu stepVec, negStepVec;
	vecScale_gpu(ray.direction_vec, fStep, stepVec);
	vecScale_gpu(stepVec, -1, negStepVec);

	//	Marching!
	//
	int count = 0; // total marching count
	float fTotalWeight = 0;
	float fTotalVal = 0;
	float fCurrMarchingLen = 0;


	int mark[4] = {id0, id1, id2, id3};
	while(fCurrMarchingLen <= fTotalLen)
	{
		point2point_gpu(point, stepVec, point);
		int layer = -1;
		float vol_val = getSampleValueByTrilinear( ray, point.data, VOL_X, VOL_Y, VOL_Z, elecData, idData, mark, (!bClipPlaneEnabled)?&layer:NULL, bShowPlane );

#ifdef DATA_2D	
		if(vol_val == INVALID_VOLVAL)
		{
			continue;
		}
#endif
		
		if(!bClipPlaneEnabled)
		{
			float val = vol_val;

			switch(tf_mode)
			{
			case 0:	// Average
				fTotalVal += val;
				fTotalWeight += 1;
				break;

			case 2:	//	Hermite mode
				float fHmtFactor = getHermiteValue(fCurrMarchingLen/fTotalLen, fP0_val, fP0_der, fP1_val, fP1_der);
				fTotalWeight += fHmtFactor;
				fTotalVal += val * fHmtFactor;
				break;
			};

		}
		else	//!bClipPlaneEnabled
		{
			float currPlaneCtr[3] = {planeCtr0, planeCtr1, planeCtr2};
			
			vect3d_gpu planeVec(planeNorm0, planeNorm1, planeNorm2);
			normalize_gpu(planeVec);

			count = 1;
			fTotalWeight = 1;

			if(point2plane_gpu(point.data, currPlaneCtr, planeVec) < 1)
			{
				fTotalVal = vol_val;
				break;
			}
		}// if(!bClipPlaneEnabled)

		fCurrMarchingLen += fStep;
		count ++;
	}//	while

	// Eliminate the weird color on the volume edges
	*ret = (!bClipPlaneEnabled && count < 10) ? 0 : fTotalVal / (bClipPlaneEnabled ? 1 : fTotalWeight);

	*nCount = count;
	return fTotalWeight;
}

#include "nanorod.cu"

__global__
void _computePixels_GPU(float *pDeviceFilm, PrimGpuObj *gpuObjs, unsigned nHeight, unsigned nWidth, 
						float xmin, float xmax,
						float ymin, float ymax,
						float zmin, float zmax,
						float fViewPlaneRatio,
						float eye_x, float eye_y, float eye_z, 
						float viewPlaneCtr_x, float viewPlaneCtr_y, float viewPlaneCtr_z, 
						float _rightVec_x, float _rightVec_y, float _rightVec_z, 
						float _upVec_x, float _upVec_y, float _upVec_z,
						unsigned max_x, unsigned max_y, unsigned max_z,
						float *elecData, int tf_mode, float fP0_val, float fP0_der, float fP1_val, float fP1_der,
						int nMultiSampleCount, float fSamplingDeltaFactor, float *rdmData, unsigned rdmCount, int bShowGeo,
						bool bClipPlaneEnabled, float planeCtr0, float planeCtr1, float planeCtr2, float planeNorm0, float planeNorm1, float planeNorm2,						
						 PrimGpuObj *pNanoDevice, unsigned nTriCount, float fNanoAlpha, 
						 int *idData, int id0, int id1, int id2, int id3, int bOnlyInRod,
						 int mMode, int bShowSlice, int bShowPlane, float fPlaneAlpha)
{

	unsigned tid = getThreadInx();

	if(tid < nHeight * nWidth)
	{
		float *pCurrPix = pDeviceFilm + tid * 3;
		
		Ray_gpu primeRay;
		float primeRdm[2] = {0, 0};

		genViewRay(primeRay, tid / nWidth, tid % nWidth, nWidth, nHeight,
						fViewPlaneRatio, 
						eye_x, eye_y, eye_z, 
						viewPlaneCtr_x, viewPlaneCtr_y, viewPlaneCtr_z, 
						_rightVec_x, _rightVec_y, _rightVec_z, 
						_upVec_x, _upVec_y, _upVec_z, primeRdm, fSamplingDeltaFactor);

		bool bHitSlice = false;
		float fSliceValue = 0;
		bool bHitPlane = false;
		float fPlaneValue = 0;
		float toolDepth = 1.f; //%%%for blending the tool
		float toolColor[3]={0.75390625,0.2109375,0.23828125};
		//	BBox for Volume
		if( isHitOnPlane(primeRay, xmin, xmax, X_AXIS) &&
			isHitOnPlane(primeRay, ymin, ymax, Y_AXIS) && 
			isHitOnPlane(primeRay, zmin, zmax, Z_AXIS) ) 
		{
	
			//	Voxels
			//
			float delta[4] = {0}; // 1st -> 2nd, x -> y
			PrimGpuObj *pObjs[2] = {0};
			float hitPoints[4][3] = {0};//%%%hitPoints can be 4 now with the little cube

			float voxColor[3] = {0};
			float sliceColor[3] = {0};
			bool toolHit;
			unsigned nHit = isHit_gpu(gpuObjs, &primeRay, pObjs, delta, hitPoints, &toolHit);
			if(nHit > 0)  
			{ 
				int inx0[3] = {0};				
				getCellInx(pObjs[0], gpuObjs, hitPoints[0], 
								max_x, max_y, max_z,
								inx0);
				
				float fCount = 0;
				int bInGeo = 0;

				int max[3] = {VOL_X, VOL_Y, VOL_Z};
				int mark[4] = {id0, id1, id2, id3};

				float value = 0;

				if(toolHit)//%%%then the little cube got hit
				{
					//get depth of center of tool
					float depth;
					depth = VOL_Y/2 - gpuObjs[6]._vCenter.data[1];
					toolDepth = depth;

					//normalize
					toolDepth /= (float)(VOL_Y);
				}
				if(nHit == 2)
				{
					//Only necessary when using dda_ray_casting
					int inx1[3] = {0};	
					getCellInx(pObjs[1], gpuObjs, hitPoints[1], 
								max_x, max_y, max_z,
								inx1);

#if 0
					fCount = dda_ray_casting(	primeRay, &value, inx0, inx1, maxes, elecData, tf_mode, fP0_val, fP0_der, fP1_val, fP1_der, bShowGeo, bInGeo,
												bClipPlaneEnabled, planeCtr0, planeCtr1, planeCtr2, planeNorm0, planeNorm1, planeNorm2,
												idData, id0, id1, id2, id3 );
#else
					int marchCount = 0;
					if(!bClipPlaneEnabled)
					{
						
						//	Show slice
						//
						if(bShowSlice)
						{

							for(int i = 0; i < nMultiSampleCount; i ++)	/// TODO: Multi-sample for only Nanorod
							{
								Ray_gpu ray;
								unsigned rdmInx = tid * 2 * nMultiSampleCount % rdmCount;
								float rdm[2] = {rdmData[rdmInx + i * 2], rdmData[rdmInx + i * 2 + 1]};

								genViewRay(ray, tid / nWidth, tid % nWidth, nWidth, nHeight,
												fViewPlaneRatio, 
												eye_x, eye_y, eye_z, 
												viewPlaneCtr_x, viewPlaneCtr_y, viewPlaneCtr_z, 
												_rightVec_x, _rightVec_y, _rightVec_z, 
												_upVec_x, _upVec_y, _upVec_z, rdm, fSamplingDeltaFactor);
								
								float sliceT = 0;
								vect3d_gpu sliceNorm;
								PrimGpuObj *pHitTri = NULL;
								if( pHitTri = isRodHit_gpu( pSliceDevice_dev, nSliceTriCount_dev, &ray, &sliceT, sliceNorm ))
								{
									bHitSlice = true;

									vect3d_gpu hitPoint;
									vect3d_gpu marchVec;
									vecScale_gpu(ray.direction_vec, sliceT, marchVec);
									point2point_gpu(ray.start_point, marchVec, hitPoint);

								fSliceValue += getSampleValueByTrilinear( ray, hitPoint.data, VOL_X, VOL_Y, VOL_Z, elecData, idData, mark, NULL, bShowPlane != 0 );
								}		
							}//	for

							fSliceValue /= nMultiSampleCount;
							custom_color_map(fSliceValue, sliceColor, gpuKnotValues, gpuKnotColors, mMode, tex_data, tex_Width, tex_Height, tex_fStart, tex_fEnd);

						}//	if(bShowSlice)
						
						//	Show Plane
						//
						if(bShowPlane)
						{

							fPlaneValue  = 0;
							for(int i = 0; i < nMultiSampleCount; i ++)	/// TODO: Multi-sample for only Nanorod
							{
								Ray_gpu ray;
								unsigned rdmInx = tid * 2 * nMultiSampleCount % rdmCount;
								float rdm[2] = {rdmData[rdmInx + i * 2], rdmData[rdmInx + i * 2 + 1]};

								genViewRay(ray, tid / nWidth, tid % nWidth, nWidth, nHeight,
												fViewPlaneRatio, 
												eye_x, eye_y, eye_z, 
												viewPlaneCtr_x, viewPlaneCtr_y, viewPlaneCtr_z, 
												_rightVec_x, _rightVec_y, _rightVec_z, 
												_upVec_x, _upVec_y, _upVec_z, rdm, fSamplingDeltaFactor);
								
								float planeT = 0;
								vect3d_gpu planeNorm;
								PrimGpuObj *pHitTri = NULL;
								if( pHitTri = isRodHit_gpu( pPlaneDevice_dev, nPlaneTriCount_dev, &ray, &planeT, planeNorm ))
								{
									bHitPlane = true;

									vect3d_gpu hitPoint;
									vect3d_gpu marchVec;
									vecScale_gpu(ray.direction_vec, planeT, marchVec);
									point2point_gpu(ray.start_point, marchVec, hitPoint);
//#ifndef DATA_2D
									fPlaneValue += getSampleValueByTrilinear( ray, hitPoint.data, VOL_X, VOL_Y, VOL_Z, elecData, idData, mark, NULL, true );
//#else
//									fPlaneValue += getSampleValueByBilinear(hitPoint.data, DIM_X, DIM_Y, VOL_Z, elecData, idData, mark);
//#endif
								}	
							}//	for

							fPlaneValue = fPlaneValue / nMultiSampleCount;

						}//	if(bShowPlane)
						//If nothing is selected, and the planes are hit, just do this
						fCount = ray_marching(	primeRay, &value, 
												&marchCount, hitPoints[0], hitPoints[1], max, elecData, 
												tf_mode, fP0_val, fP0_der, fP1_val, fP1_der, bShowGeo, bInGeo,
												bClipPlaneEnabled, planeCtr0, planeCtr1, planeCtr2, planeNorm0, planeNorm1, planeNorm2,
												idData, id0, id1, id2, id3, bShowSlice, bShowPlane);
					}
					else	//if(!bClipPlaneEnabled)
					{
						float totalVal = 0;

						for(int i = 0; i < nMultiSampleCount; i ++)	/// TODO: Multi-sample for only Nanorod
						{
							Ray_gpu ray;
							unsigned rdmInx = tid * 2 * nMultiSampleCount % rdmCount;
							float rdm[2] = {rdmData[rdmInx + i * 2], rdmData[rdmInx + i * 2 + 1]};

							genViewRay(ray, tid / nWidth, tid % nWidth, nWidth, nHeight,
											fViewPlaneRatio, 
											eye_x, eye_y, eye_z, 
											viewPlaneCtr_x, viewPlaneCtr_y, viewPlaneCtr_z, 
											_rightVec_x, _rightVec_y, _rightVec_z, 
											_upVec_x, _upVec_y, _upVec_z, rdm, fSamplingDeltaFactor);

							float tmpVal = 0;
							fCount = ray_marching(	ray, &tmpVal, 
												&marchCount, hitPoints[0], hitPoints[1], max, elecData, 
												tf_mode, fP0_val, fP0_der, fP1_val, fP1_der, bShowGeo, bInGeo,
												bClipPlaneEnabled, planeCtr0, planeCtr1, planeCtr2, planeNorm0, planeNorm1, planeNorm2,
												idData, id0, id1, id2, id3, bShowSlice );

							totalVal += tmpVal;
						}//for

						value = totalVal / nMultiSampleCount;
					}//if(!bClipPlaneEnabled)
#endif
				}// if(nHit == 2)

				//	Customed Color-map		
				//

				custom_color_map(	value, voxColor, gpuKnotValues, gpuKnotColors, mMode, tex_data, tex_Width, tex_Height, tex_fStart, tex_fEnd);
			}//	hit or not
			else
			{
				voxColor[0] = AMBI_X;
				voxColor[1] = AMBI_Y;
				voxColor[2] = AMBI_Z;
			}
			
			//	Nanorod
			//
			vect3d_gpu nanoColor;	
			bool bHitNano = false;
			if(bHitSlice) bHitNano = true;
			if(bShowGeo && !bHitSlice)
			{
				for(int i = 0; i < nMultiSampleCount; i ++)	/// TODO: Multi-sample for only Nanorod
				{
					Ray_gpu ray;
					unsigned rdmInx = tid * 2 * nMultiSampleCount % rdmCount;
					float rdm[2] = {rdmData[rdmInx + i * 2], rdmData[rdmInx + i * 2 + 1]};

					genViewRay(ray, tid / nWidth, tid % nWidth, nWidth, nHeight,
									fViewPlaneRatio, 
									eye_x, eye_y, eye_z, 
									viewPlaneCtr_x, viewPlaneCtr_y, viewPlaneCtr_z, 
									_rightVec_x, _rightVec_y, _rightVec_z, 
									_upVec_x, _upVec_y, _upVec_z, rdm, fSamplingDeltaFactor);
					
					float nanoT = 0;
					vect3d_gpu norm;
					PrimGpuObj *pHitTri = NULL;
					if( pHitTri = isRodHit_gpu( pNanoDevice, nTriCount, &ray, &nanoT, norm ))
					{
						vect3d_gpu lightPos(0, 200, 0);

						//	Cap 0
						//
						bool bHitCap0 = false;
						vect3d_gpu cap0Color;
						float cap0T = 0;
						vect3d_gpu normCap0;
						PrimGpuObj *pHitCap0 = isRodHit_gpu( pCap0Device_dev, nCap0TriCount_dev, &ray, &cap0T, normCap0 );
						if(pHitCap0)
						{
							vect3d_gpu tmpColor;

							vect3d_gpu marchVec;
							vecScale_gpu(ray.direction_vec, cap0T, marchVec);
							vect3d_gpu hitPoint;
							point2point_gpu(ray.start_point, marchVec, hitPoint);

							if(!bShowPlane ||  bShowPlane && hitPoint[1] < 0)
							{
								evalPhong(ray.start_point, hitPoint, normCap0, pHitCap0, lightPos, tmpColor);
								point2point_gpu(cap0Color, tmpColor, cap0Color);

								bHitCap0 = true;
							}
						}

						//	Cap 1
						//
						bool bHitCap1 = false;
						vect3d_gpu cap1Color;
						float cap1T = 0;
						vect3d_gpu normCap1;
						PrimGpuObj *pHitCap1 = isRodHit_gpu( pCap1Device_dev, nCap1TriCount_dev, &ray, &cap1T, normCap1 );
						if(pHitCap1)
						{
							vect3d_gpu tmpColor;

							vect3d_gpu marchVec;
							vecScale_gpu(ray.direction_vec, cap1T, marchVec);
							vect3d_gpu hitPoint;
							point2point_gpu(ray.start_point, marchVec, hitPoint);

							if(!bShowPlane ||  bShowPlane && hitPoint[1] < 0)
							{
								evalPhong(ray.start_point, hitPoint, normCap1, pHitCap1, lightPos, tmpColor);
								point2point_gpu(cap1Color, tmpColor, cap1Color);

								bHitCap1 = true;
							}
						}
						
						//	Rod itself
						//
						bool bHitRod = false;
						vect3d_gpu tmpColor;

						vect3d_gpu marchVec;
						vecScale_gpu(ray.direction_vec, nanoT, marchVec);
						vect3d_gpu hitPoint;
						point2point_gpu(ray.start_point, marchVec, hitPoint);

						if(!bShowPlane ||  bShowPlane && hitPoint[1] < 0)
						{
							evalPhong(ray.start_point, hitPoint, norm, pHitTri, lightPos, tmpColor);
							point2point_gpu(nanoColor, tmpColor, nanoColor);

							bHitRod = true;
						}

						//	Blend Rod\Cap0\Cap1
						if(tf_mode == 1)
						{
							vect3d_gpu lightGreen(212.f/255.f, 231.f/255.f, 178.f/255.f);
							vect3d_gpu   midGreen(167.f/255.f, 214.f/255.f, 148.f/255.f);
							vect3d_gpu darkGreen(153.f/255.f, 183.f/255.f,  74.f/255.f);
							//vect3d_gpu darkGreen(0.2, 0.2, 0.2);

							if(bHitCap1) vecScale_gpu(darkGreen, vecLen_gpu(&cap1Color), cap1Color);
							if(bHitCap0) vecScale_gpu(midGreen,  vecLen_gpu(&cap0Color), cap0Color);
							vecScale_gpu(lightGreen, vecLen_gpu(&nanoColor), nanoColor);
						}

						if(tf_mode == 1)
						{
							//float alpha0 = 0.8;
							float alphaNano = 0.25;
							float alphaCap0 = 0.6;
							float alphaCap1 = 0.7;

							////	blend
							//
							vecScale_gpu(cap0Color, alphaCap0, cap0Color);
							vecScale_gpu(cap1Color, alphaCap1, cap1Color);
							vecScale_gpu(nanoColor, alphaNano, nanoColor);

							point2point_gpu(nanoColor, cap0Color, nanoColor);
							point2point_gpu(nanoColor, cap1Color, nanoColor);

							for(int i = 0; i < 3; i ++)
							{
								nanoColor.data[i] = nanoColor.data[i] > 1 ? 1 : nanoColor.data[i];
								nanoColor.data[i] = nanoColor.data[i] < 0 ? 0 : nanoColor.data[i];
							}
							bHitNano = true;
						}
						else if( tf_mode != -1 && (bHitRod || bHitPlane) )
						{
							vecScale_gpu(cap1Color, fNanoAlpha, cap1Color);
							vecScale_gpu(cap0Color, 1 - fNanoAlpha, cap0Color);
							point2point_gpu(cap0Color, cap1Color, cap0Color);

							vecScale_gpu(nanoColor, fNanoAlpha, nanoColor);
							vecScale_gpu(cap0Color, 1 - fNanoAlpha, cap0Color);
							point2point_gpu(nanoColor, cap0Color, nanoColor);

							for(int i = 0; i < 3; i ++)
							{
								nanoColor.data[i] = nanoColor.data[i] > 1 ? 1 : nanoColor.data[i];
								nanoColor.data[i] = nanoColor.data[i] < 0 ? 0 : nanoColor.data[i];
							}
							bHitNano = true;
						}
						
					}			
					else
					{
						if(bOnlyInRod)
						{
							nanoColor.data[0] += AMBI_X;
							nanoColor.data[1] += AMBI_Y;
							nanoColor.data[2] += AMBI_Z;
						}
						else
						{
							nanoColor.data[0] += voxColor[0];
							nanoColor.data[1] += voxColor[1];
							nanoColor.data[2] += voxColor[2];
						}
					}
				}//	for

				vecScale_gpu(nanoColor, 1.f / nMultiSampleCount, nanoColor);
			}//	if(bShowgeo)

			if(bHitNano)
			{
				if(tf_mode == 1)
				{
					*(pCurrPix + 0) = nanoColor.data[0];
					*(pCurrPix + 1) = nanoColor.data[1];
					*(pCurrPix + 2) = nanoColor.data[2];

				}
				else
				{
					if(bHitSlice)
					{
						float tmp1 = 0.5;
						*(pCurrPix + 0) = voxColor[0] * (1 - tmp1) + sliceColor[0] * tmp1;
						*(pCurrPix + 1) = voxColor[1] * (1 - tmp1) + sliceColor[1] * tmp1;
						*(pCurrPix + 2) = voxColor[2] * (1 - tmp1) + sliceColor[2] * tmp1;
					}
					else
					{
						*(pCurrPix + 0) = voxColor[0] * (1.0 - fNanoAlpha) + nanoColor.data[0] * fNanoAlpha;
						*(pCurrPix + 1) = voxColor[1] * (1.0 - fNanoAlpha) + nanoColor.data[1] * fNanoAlpha;
						*(pCurrPix + 2) = voxColor[2] * (1.0 - fNanoAlpha) + nanoColor.data[2] * fNanoAlpha;
					}

					//	blend plane
					if(bShowPlane && bHitPlane)
					{
						float planeColor[3] = {0};
#ifndef DATA_2D
						fPlaneValue *= 3;
#endif
						custom_color_map( fPlaneValue, planeColor, gpuKnotValues, gpuKnotColors, mMode, tex_data, tex_Width, tex_Height, tex_fStart, tex_fEnd);
						//
						float t = 0.5;
						*(pCurrPix + 0) = planeColor[0] * t + voxColor[0] * (1.0 - t);
						*(pCurrPix + 1) = planeColor[1] * t + voxColor[1] * (1.0 - t);
						*(pCurrPix + 2) = planeColor[2] * t + voxColor[2] * (1.0 - t);
					}
					else if(bShowPlane && !bHitPlane)
					{
						*(pCurrPix + 0) = voxColor[0] * (fNanoAlpha)/* + nanoColor.data[0] * (1 - fNanoAlpha)*/;
						*(pCurrPix + 1) = voxColor[1] * (fNanoAlpha)/* + nanoColor.data[1] * (1 - fNanoAlpha)*/;
						*(pCurrPix + 2) = voxColor[2] * (fNanoAlpha)/* + nanoColor.data[2] * (1 - fNanoAlpha)*/;
					}
				}
				//%%%blend with tool
				*(pCurrPix + 0) = toolColor[0] * (1.0 - toolDepth) + *(pCurrPix + 0) * toolDepth;
				*(pCurrPix + 1) = toolColor[1] * (1.0 - toolDepth) + *(pCurrPix + 1) * toolDepth;
				*(pCurrPix + 2) = toolColor[2] * (1.0 - toolDepth) + *(pCurrPix + 2) * toolDepth;
			}// hitNano
			else
			{
				if(!bOnlyInRod)
				{
					*(pCurrPix + 0) = 1.f * 0.5f + (0.5f)*voxColor[0];
					*(pCurrPix + 1) = 1.f * 0.5f + (0.5f)*voxColor[1];
					*(pCurrPix + 2) = 1.f * 0.5f + (0.5f)*voxColor[2];

					//	blend plane
					if(bShowPlane && bHitPlane)
					{
						float planeColor[3] = {0};
						
						custom_color_map( fPlaneValue, planeColor, gpuKnotValues, gpuKnotColors, mMode, tex_data, tex_Width, tex_Height, tex_fStart, tex_fEnd);
						//
#ifndef DATA_2D
						*(pCurrPix + 0) = planeColor[0]/* * fNanoAlpha + voxColor[0] * (1.0 - fNanoAlpha)*/;
						*(pCurrPix + 1) = planeColor[1]/* * fNanoAlpha + voxColor[1] * (1.0 - fNanoAlpha)*/;
						*(pCurrPix + 2) = planeColor[2]/* * fNanoAlpha + voxColor[2] * (1.0 - fNanoAlpha)*/;
#else
						*(pCurrPix + 0) = planeColor[0] * fNanoAlpha + voxColor[0] * (1.0 - fNanoAlpha);
						*(pCurrPix + 1) = planeColor[1] * fNanoAlpha + voxColor[1] * (1.0 - fNanoAlpha);
						*(pCurrPix + 2) = planeColor[2] * fNanoAlpha + voxColor[2] * (1.0 - fNanoAlpha);
#endif
					}
				}
				else
				{
					*(pCurrPix + 0) = AMBI_X;
					*(pCurrPix + 1) = AMBI_Y;
					*(pCurrPix + 2) = AMBI_Z;
				}
				//%%%blend with tool
				*(pCurrPix + 0) = toolColor[0] * (1.0 - toolDepth) + *(pCurrPix + 0) * toolDepth;
				*(pCurrPix + 1) = toolColor[1] * (1.0 - toolDepth) + *(pCurrPix + 1) * toolDepth;
				*(pCurrPix + 2) = toolColor[2] * (1.0 - toolDepth) + *(pCurrPix + 2) * toolDepth;
			}
		}// if bbox
		else
		{
			//	Not hit
			*(pCurrPix + 0) = AMBI_X;
			*(pCurrPix + 1) = AMBI_Y;
			*(pCurrPix + 2) = AMBI_Z;
			//%%%blend with tool
			*(pCurrPix + 0) = toolColor[0] * (1.0 - toolDepth) + *(pCurrPix + 0) * toolDepth;
			*(pCurrPix + 1) = toolColor[1] * (1.0 - toolDepth) + *(pCurrPix + 1) * toolDepth;
			*(pCurrPix + 2) = toolColor[2] * (1.0 - toolDepth) + *(pCurrPix + 2) * toolDepth;
		}
	}// if tid
}

////
void Tracer::setVolBBox(	float xmin, float xmax,
							float ymin, float ymax, 
							float zmin, float zmax)
{
	_bbox.setDim( xmin, xmax,
				  ymin, ymax,
				  zmin, zmax);
}

void Tracer::computePixels_GPU(float *pDeviceFilm, unsigned nHeight, unsigned nWidth,
								   float fViewPlaneRatio,
								   float eye_x, float eye_y, float eye_z, 
									float viewPlaneCtr_x, float viewPlaneCtr_y, float viewPlaneCtr_z, 
									float _rightVec_x, float _rightVec_y, float _rightVec_z, 
									float _upVec_x, float _upVec_y, float _upVec_z,
									unsigned max_x, unsigned max_y, unsigned max_z,
									float *elecData, int tf_mode, float fP0_val, float fP0_der, float fP1_val, float fP1_der,
									int nMultiSampleCount, float fSamplingDeltaFactor, float *rdmData, unsigned rdmCount, int bShowGeo,
						bool bClipPlaneEnabled, float planeCtr0, float planeCtr1, float planeCtr2, float planeNorm0, float planeNorm1, float planeNorm2,
						float knotValue0, float knotValue1, float knotValue2, float knotValue3, float knotValue4, 
						int knotColor0, int knotColor1, int knotColor2, int knotColor3, int knotColor4,
						 PrimGpuObj *pNanoDevice, unsigned nTriCount, float fNanoAlpha, int *idData, int mark[4], int bOnlyInRod,
						 int mMode, float *deviceTexData, int texWidth, int texHeight, float fStart, float fEnd)
{

	_setTexInfo<<<1, 1>>>(fStart, fEnd, texWidth, texHeight, deviceTexData, nPlaneSampleCount,
							pCap0Device, nCap0TriCount, pCap1Device, nCap1TriCount,
							pSliceDevice, nSliceTriCount, 
							knotValue0, knotValue1, knotValue2, knotValue3, knotValue4, 
							knotColor0, knotColor1, knotColor2, knotColor3, knotColor4,
							pNanoPlaneDevice, nPlaneTriCount);
	hipDeviceSynchronize();

	unsigned nTotalPixel = nHeight * nWidth;	//	As below, yes, the resolution has to be 256x
	_computePixels_GPU<<<nTotalPixel / GpuBlockSize, GpuBlockSize>>>(pDeviceFilm, gpuObjs, nHeight, nWidth, 
											_bbox._xmin, _bbox._xmax, 
											_bbox._ymin, _bbox._ymax,
											_bbox._zmin, _bbox._zmax,
											fViewPlaneRatio,
											eye_x, eye_y, eye_z, 
											viewPlaneCtr_x, viewPlaneCtr_y, viewPlaneCtr_z, 
											_rightVec_x, _rightVec_y, _rightVec_z, 
											_upVec_x, _upVec_y, _upVec_z,
											max_x, max_y, max_z,
											elecData, tf_mode, fP0_val, fP0_der, fP1_val, fP1_der, 
											nMultiSampleCount, fSamplingDeltaFactor, rdmData, rdmCount, bShowGeo,
											bClipPlaneEnabled, planeCtr0, planeCtr1, planeCtr2, planeNorm0, planeNorm1, planeNorm2,
											pNanoDevice, nTriCount, fNanoAlpha, idData, mark[0], mark[1], mark[2], mark[3], bOnlyInRod,
											mMode, bShowSlice, bShowPlane, fPlaneAlpha); 
	hipDeviceSynchronize();

	//if(bClipPlaneEnabled)
	//{
	//	bilinear_convolution<<<nTotalPixel / GpuBlockSize, GpuBlockSize>>>(pDeviceFilm, nHeight, nWidth, nPlaneSampleCount);
	//	hipDeviceSynchronize();
	//}
}
